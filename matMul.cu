#include <stdio.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define NUM_NEURONS 1024
#define NUM_IMAGES 60000
#define NUM_LAYERS 1
__global__ void simpleMatMulKernel(short * image, float * weights, float * results) {
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    for(int i = 0; i < 1024 ; i ++) {
        results[index] += image[i]*weights[i + NUM_NEURONS*index];
    }
    // printf("%d\n", index);
    return;
}

void simplemutMulCuda(short *image, float weights[][1024], float *results) {

    short* device_image;
    float* device_wts;
    float* device_results;

    hipMalloc((void**) &device_image, NUM_NEURONS * sizeof(short));
    hipMalloc((void**) &device_wts, NUM_LAYERS * NUM_NEURONS * NUM_NEURONS * sizeof(float));
    hipMalloc((void**) &device_results, NUM_NEURONS * sizeof(float));
    
    // dim3 blockDim(512, 1);
    // dim3 gridDim(2);

    hipMemcpy(device_image, image, NUM_NEURONS * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(device_wts, weights, NUM_LAYERS * NUM_NEURONS * NUM_NEURONS * sizeof(float), hipMemcpyHostToDevice);

    simpleMatMulKernel<<<32, 32>>>(device_image, device_wts, device_results);

    hipMemcpy(results, device_results,  NUM_NEURONS * sizeof(float), hipMemcpyDeviceToHost);

}